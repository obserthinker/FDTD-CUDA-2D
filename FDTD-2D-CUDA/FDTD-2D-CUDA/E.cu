#include "E.cuh"

//using namespace std;

const float epsilon = 8.85e-12f;

E::E(src source)
{
	Ez_init(source);
	coe_Ez_set(source);
	Ez_boundary_init(source);
}

void E::coe_Ez_set(src source)
{
	coe_Ez = source.dt / (epsilon * source.dz);
}

void E::Ez_init(src source)
{
	int i, j;
	//initialize Ez
	size_Ez_x = source.size_space_x + 1;
	size_Ez_y = source.size_space_y + 1;
	size_Ez = size_Ez_x * size_Ez_y;

	Ez = (float*)malloc(size_Ez * sizeof(float));
	hipMallocPitch(&dev_Ez, &pitch_Ez, size_Ez_x * sizeof(float), size_Ez_y);
	ele_Ez = pitch_Ez / sizeof(float);

	for ( i = 0; i < size_Ez_y; i++){
		for (j = 0; j < size_Ez_x; j++){
			Ez[i * size_Ez_x + j] = 0.f;
		}
	}

	//initialize file
	fstream myfile;
	myfile.open("Ez.txt", ios::out);
	myfile.close();

	//initialize boundary
	Ez_boundary_init(source);
}

void E::Ez_boundary_init(src source)
{
	hipMalloc(&E_bd_l, size_Ez_y * sizeof(float));
	hipMalloc(&E_bd_r, size_Ez_y * sizeof(float));
	hipMalloc(&E_bd_u, size_Ez_x * sizeof(float));
	hipMalloc(&E_bd_d, size_Ez_x * sizeof(float));
	hipMalloc(&E_nbd_l, size_Ez_y * sizeof(float));
	hipMalloc(&E_nbd_r, size_Ez_y * sizeof(float));
	hipMalloc(&E_nbd_u, size_Ez_x * sizeof(float));
	hipMalloc(&E_nbd_d, size_Ez_x * sizeof(float));
	hipMemset(E_bd_l, 0, size_Ez_y * sizeof(float));
	hipMemset(E_bd_r, 0, size_Ez_y * sizeof(float));
	hipMemset(E_bd_u, 0, size_Ez_x * sizeof(float));
	hipMemset(E_bd_d, 0, size_Ez_x * sizeof(float));
	hipMemset(E_nbd_l, 0, size_Ez_y * sizeof(float));
	hipMemset(E_nbd_r, 0, size_Ez_y * sizeof(float));
	hipMemset(E_nbd_u, 0, size_Ez_x * sizeof(float));
	hipMemset(E_nbd_d, 0, size_Ez_x * sizeof(float));

	coe_MUR = (source.C * source.dt - source.dz) / (source.C * source.dt + source.dz);
}

void E::Ez_checkout()
{
	int i, j;
	cout << "Ez size: " << size_Ez << endl;
	for (i = 0; i < size_Ez_y; i++){
		for (j = 0; j < size_Ez_x; j++){
			cout << Ez[i * size_Ez_x + j] << "\t";
		}
		cout << endl;
	}
	cout << endl;
}

void E::Ez_save2file()
{
	int i, j;
	fstream myfile;
	myfile.open("Ez.txt", ios::app);

	for (i = 0; i < size_Ez_y; i++){
		for (j = 0; j < size_Ez_x; j++){
			myfile << Ez[i * size_Ez_x + j] << "\t";
		}
		myfile << endl;
	}
	myfile << endl;
	myfile.close();
}
