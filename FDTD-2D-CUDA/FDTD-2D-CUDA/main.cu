#include "hip/hip_runtime.h"
#include "E.cuh"
#include "H.cuh"
#include "src.cuh"
#include "cmp.cuh"
#include <iostream>

using namespace std;

int main()
{
	src source(30,30,100);
	H Hxy(source);
	E Ez(source);
	size_t pitch_Hx, pitch_Hy, pitch_Ez;
	float *dev_Hx, *dev_Hy, *dev_Ez;
	int i, ele_ex, ele_hx, ele_hy;
	hipError_t se, shx, shy, err;

	source.src_checkout();

	shx = hipMallocPitch(&dev_Hx, &pitch_Hx, Hxy.size_Hx_x * sizeof(float), Hxy.size_Hx_y);
	shy = hipMallocPitch(&dev_Hy, &pitch_Hy, Hxy.size_Hy_x * sizeof(float), Hxy.size_Hy_y);
	se = hipMallocPitch(&dev_Ez, &pitch_Ez, Ez.size_Ez_x * sizeof(float), Ez.size_Ez_y);
	ele_ex = pitch_Ez / sizeof(float);
	ele_hx = pitch_Hx / sizeof(float);
	ele_hy = pitch_Hy / sizeof(float);

	if (shx == hipSuccess && shy == hipSuccess && se == hipSuccess){
		for (i = 0; i < source.size_time; i++){
			err = hipMemcpy2D(dev_Ez, pitch_Ez, Ez.Ez, Ez.size_Ez_x*sizeof(float), Ez.size_Ez_x*sizeof(float), Ez.size_Ez_y, hipMemcpyHostToDevice);
			Hy_cmp_kernel<<<1,1>>>(dev_Ez, dev_Hy, Hxy.size_Hy_x, Hxy.size_Hy_y, Hxy.coe_H, ele_ex, ele_hy);
			Hx_cmp_kernel<<<1,1>>>(dev_Ez, dev_Hx, Hxy.size_Hx_x, Hxy.size_Hx_y, Hxy.coe_H, ele_ex, ele_hy);
			Ez_cmp_kernel<<<1,1>>>(dev_Ez, dev_Hx, dev_Hy, Ez.coe_Ez, Ez.size_Ez_x, Ez.size_Ez_y, ele_ex, ele_hx, ele_hy);
			Ez_boundary_PEC<<<1,1>>>(dev_Ez, Ez.size_Ez_x, Ez.size_Ez_y, ele_ex);
			src_cmp_kernel<<<1,1>>>(i, dev_Ez, Ez.size_Ez_x, Ez.size_Ez_y, source.dt, ele_ex);
			hipMemcpy2D(Ez.Ez, Ez.size_Ez_x*sizeof(float), dev_Ez, pitch_Ez, Ez.size_Ez_x*sizeof(float), Ez.size_Ez_y, hipMemcpyDeviceToHost);
			Ez.Ez_save2file();
		}
	}
	else{
		cout << "hipMalloc failed" << endl;
		return 0;
	}
	return 0;
}