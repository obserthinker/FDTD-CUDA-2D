#include "hip/hip_runtime.h"
#include "E.cuh"
#include "H.cuh"
#include "src.cuh"
#include "cmp.cuh"
#include <iostream>

using namespace std;

int main()
{
	src source(100, 100, 300);
	H Hxy(source);
	E Ez(source);
	int i;

	source.src_checkout();

	cout << Ez.ele_Ez << "\t" << Ez.size_Ez_x << endl;
	for (i = 0; i < source.size_time; i++){
		//hipMemcpy2D(Ez.dev_Ez, Ez.pitch_Ez, Ez.Ez, Ez.size_Ez_x*sizeof(float), Ez.size_Ez_x*sizeof(float), Ez.size_Ez_y, hipMemcpyHostToDevice);

		Hy_cmp_kernel << <50, 50 >> >(Ez.dev_Ez, Hxy.dev_Hy, Hxy.size_Hy_x, Hxy.size_Hy_y, Hxy.coe_H, Ez.ele_Ez, Hxy.ele_Hy);
		Hx_cmp_kernel << <50, 50 >> >(Ez.dev_Ez, Hxy.dev_Hx, Hxy.size_Hx_x, Hxy.size_Hx_y, Hxy.coe_H, Ez.ele_Ez, Hxy.ele_Hy);
		Ez_cmp_kernel << <50, 50 >> >(Ez.dev_Ez, Hxy.dev_Hx, Hxy.dev_Hy, Ez.coe_Ez, Ez.size_Ez_x, Ez.size_Ez_y, Ez.ele_Ez, Hxy.ele_Hx, Hxy.ele_Hy);
		//Ez_boundary_PEC<<<1,1>>>(Ez.dev_Ez, Ez.size_Ez_x, Ez.size_Ez_y, Ez.ele_Ez);
		Ez_MUR_u << <50, 50 >> >(Ez.dev_Ez, Ez.E_bd_u, Ez.E_nbd_u, Ez.size_Ez_x, Ez.size_Ez_y, Ez.coe_MUR, Ez.ele_Ez);
		Ez_MUR_d << <50, 50 >> >(Ez.dev_Ez, Ez.E_bd_d, Ez.E_nbd_d, Ez.size_Ez_x, Ez.size_Ez_y, Ez.coe_MUR, Ez.ele_Ez);
		Ez_MUR_lr << <1, Ez.size_Ez_y >> >(Ez.dev_Ez, Ez.E_bd_l, Ez.E_nbd_l, Ez.E_bd_r, Ez.E_nbd_r, Ez.size_Ez_x, Ez.size_Ez_y, Ez.coe_MUR, Ez.ele_Ez);
		src_cmp_kernel << <1, 1 >> >(i, Ez.dev_Ez, Ez.size_Ez_x, Ez.size_Ez_y, source.dt, Ez.ele_Ez);

		hipMemcpy2D(Ez.Ez, Ez.size_Ez_x*sizeof(float), Ez.dev_Ez, Ez.pitch_Ez, Ez.size_Ez_x*sizeof(float), Ez.size_Ez_y, hipMemcpyDeviceToHost);
		Ez.Ez_save2file();
	}
	return 0;
}