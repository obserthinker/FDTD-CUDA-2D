#include "H.cuh"

H::H(src source)
{
	Hx_init(source);
	Hy_init(source);
	coe_H_set(source);
}

void H::Hx_init(src source)
{
	int i, j;
    //size_Hx = source.size_space_x * (source.size_space_y - 1);
	size_Hx_x = source.size_space_x;
	size_Hx_y = source.size_space_y + 1;
	size_Hx = size_Hx_x * size_Hx_y;

    Hx = (float*)malloc(size_Hx * sizeof(float));
	hipMallocPitch(&dev_Hx, &pitch_Hx, size_Hx_x * sizeof(float), size_Hx_y);
	ele_Hx = pitch_Hx / sizeof(float);

    for (i = 0; i < size_Hx_y; ++i){
		for ( j = 0; j < size_Hx_x; j++){
			Hx[i*size_Hx_x +j] = 0.f;
		}
    }

	fstream myfile;
	myfile.open("Hx.txt", ios::out);
	myfile.close();
}

void H::Hy_init(src source)
{
	int i, j;
    //size_Hy = (source.size_space_x - 1) * source.size_space_y;
	size_Hy_x = source.size_space_x + 1;
	size_Hy_y = source.size_space_y;
	size_Hy = size_Hy_x * size_Hy_y;

    Hy = (float*)malloc(size_Hy * sizeof(float));
	hipMallocPitch(&dev_Hy, &pitch_Hy, size_Hy_x * sizeof(float), size_Hy_y);
	ele_Hy = pitch_Hy / sizeof(float);

    for (i = 0; i < size_Hy_y; ++i){
    	for (j = 0; j < size_Hy_x; j++){
			Hy[i*size_Hy_x + j] = 0.f;
		} 
    }

	fstream myfile;
	myfile.open("Hx.txt", ios::out);
	myfile.close();
}

void H::coe_H_set(src source)
{
	coe_H = source.dt / (mu * source.dz);
}

void H::Hx_checkout()
{
	int i, j;
	cout << "Hx size: " << size_Hx << endl;
	for (i = 0; i < size_Hx_y; i++){
		for (j = 0; j < size_Hx_x; j++){
			cout << Hx[i*size_Hx_x + j] << "\t";
		}
	}
	cout << endl;
}

void H::Hy_checkout()
{
	int i, j;
	cout << "Hy size: " << size_Hy << endl;
	for (i = 0; i < size_Hy_y; i++)
	{
		for (j = 0; j < size_Hy_x; j++){
			cout << Hy[i*size_Hy_x + j] << "\t";
		}
	}
	cout << endl;
}

void H::Hx_save2file()
{
	int i, j;
	fstream myfile;
	myfile.open("Hx.txt", ios::app);

	for ( i = 0; i < size_Hx_y; i++){
		for ( j = 0; j < size_Hx_x; j++){
			myfile << Hx[i*size_Hx_x + j] << "\t";
		}
		myfile << endl;
	}
	myfile << endl;
	myfile.close();
}

void H::Hy_save2file()
{
	int i, j;
	fstream myfile;

	myfile.open("Hy.txt", ios::app);

	for (i = 0; i < size_Hy_y; i++){
		for (j = 0; j < size_Hy_x; j++){
			myfile << Hy[i*size_Hy_x + j] << "\t";
		}
		myfile << endl;
	}
	myfile << endl;
	myfile.close();
}
