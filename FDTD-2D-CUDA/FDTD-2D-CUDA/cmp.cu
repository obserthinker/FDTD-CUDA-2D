#include "hip/hip_runtime.h"
#include "cmp.cuh"

__global__ 
void src_cmp_kernel(
int current_timestep, 
float* dev_Ez, 
int size_Ez_x, int size_Ez_y, float dt, int ele_ex
)
{
	float T,T0;
	float vt, val_src, time;
	int src_pos_x, src_pos_y;

	time = current_timestep * dt;

	T = 5e-10f;
	T0 = 3 * T;
	vt = (time - T0) / T;

	val_src = expf(-powf(vt, 2.0f));

	src_pos_x = size_Ez_y / 2;
	src_pos_y = size_Ez_x / 2;

	dev_Ez[src_pos_y * ele_ex + src_pos_x] = val_src;
}

__global__ 
void Ez_cmp_kernel(
					float* Ez, float* Hx, float* Hy, 
					float coe_Ez, int size_Ez_x, int size_Ez_y, 
					int ele_ex, int ele_hx, int ele_hy
)
{
	int i, j;
	float dif_Hy, dif_Hx;
	for (i = 1; i < size_Ez_y - 1; i++){
		for (j = 1; j < size_Ez_x - 1; j++){
			//Hy(i,j)	-	Hy(i-1,j)
			dif_Hy = Hy[i*ele_hy + j] - Hy[(i - 1)* ele_hy + j];
			//Hx(i,j-1)	-	Hx(i,j)
			dif_Hx = Hx[i*ele_hx + (j - 1)] - Hx[i*ele_hx + j];
			Ez[i*ele_hx + j] += coe_Ez * (dif_Hx + dif_Hy);
		}
	}
}

__global__ void Hx_cmp_kernel(
	float* Ez, float* Hx, 
	int size_Hx_x, int size_Hx_y,
	float coe_H, int ele_ex, int ele_hx
)
{
	int i, j;

	for ( i = 0; i < size_Hx_y; i++){
		for ( j = 0; j < size_Hx_x; j++){
			Hx[i*ele_hx + j] += coe_H *
				(Ez[i*ele_ex + j] - Ez[i*ele_ex + (j + 1)]);
		}
	}
}

__global__ void Hy_cmp_kernel(
	float* Ez, float* Hy,
	int size_Hy_x, int size_Hy_y,
	float coe_H, int ele_ex, int ele_hy
)
{
	int i, j;

	for ( i = 0; i < size_Hy_y; i++){
		for ( j = 0; j < size_Hy_x; j++){
			Hy[i*ele_hy + j] += 
				coe_H * (Ez[(i + 1)*ele_ex + j] - Ez[i*ele_ex + j]);
		}
	}
}

__global__ void Ez_boundary_PEC(
	float* Ez, 
	int size_Ez_x, int size_Ez_y, int ele_ex
	)
{
	int i, j;

	for (i = 0; i < size_Ez_y; i++){
		if (i == 0 || i == (size_Ez_y - 1)){
			for (j = 0; j < size_Ez_x; j++){
				Ez[i * ele_ex + j] = 0.f;
			}
		}
		else{
			Ez[i* ele_ex + 0] = 0.f;
			Ez[i * ele_ex + size_Ez_x - 1] = 0.f;
		}
	}
}

__global__
void Ez_MUR_u(
float* Ez, float* E_bd_u, float* E_nbd_u, 
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	int i;
	for ( i = 0; i < size_Ez_x; i++){
		Ez[(size_Ez_y - 1) * ele_ex + i] = E_nbd_u[i]
			+ coe_MUR * (Ez[(size_Ez_y - 2) * ele_ex + i]
			- E_bd_u[i]);
		E_nbd_u[i] = Ez[(size_Ez_y - 2) * ele_ex + i];
		E_bd_u[i] = Ez[(size_Ez_y - 1) * ele_ex + i];
	}
}

__global__
void Ez_MUR_d(
float* Ez, float* E_bd_d, float* E_nbd_d, 
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	int i;
	for ( i = 0; i < size_Ez_x; i++){
		Ez[i] = E_nbd_d[i] + coe_MUR * (Ez[1*ele_ex + i]
			- E_bd_d[i]);
		E_nbd_d[i] = Ez[1*ele_ex + i];
		E_bd_d[i] = Ez[i];
	}
}

__global__
void Ez_MUR_lr(
float* Ez, float* E_bd_l, float* E_nbd_l, 
float* E_bd_r, float* E_nbd_r,
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	for (int i = 0; i < size_Ez_y; i++){
		//left
		Ez[i*ele_ex + 0] = E_nbd_l[i] + coe_MUR *
			(Ez[i * ele_ex + 1] - E_bd_l[i]);
		E_nbd_l[i] = Ez[i * ele_ex + 1];
		E_bd_l[i] = Ez[i * ele_ex + 0];
		//right
		Ez[i* ele_ex + (size_Ez_x - 1)] = E_nbd_r[i] + coe_MUR *
			(Ez[i * ele_ex + (size_Ez_x - 2)] - E_bd_r[i]);
		E_nbd_r[i] = Ez[i * ele_ex + (size_Ez_x - 2)];
		E_bd_r[i] = Ez[i* ele_ex + (size_Ez_x - 1)];
	}
}
