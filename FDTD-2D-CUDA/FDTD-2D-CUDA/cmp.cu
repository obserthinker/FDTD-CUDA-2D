#include "hip/hip_runtime.h"
#include "cmp.cuh"

__global__ 
void src_cmp_kernel(
int current_timestep, 
float* dev_Ez, 
int size_Ez_x, int size_Ez_y, float dt, int ele_ex
)
{
	float T,T0;
	float vt, val_src, time;
	int src_pos_x, src_pos_y;

	time = current_timestep * dt;

	T = 5e-10f;
	T0 = 3 * T;
	vt = (time - T0) / T;

	val_src = expf(-powf(vt, 2.0f));

	src_pos_x = size_Ez_y / 2;
	src_pos_y = size_Ez_x / 2;

	dev_Ez[src_pos_y * ele_ex + src_pos_x] = val_src;
}

__global__ 
void Ez_cmp_kernel(
					float* Ez, float* Hx, float* Hy, 
					float coe_Ez, int size_Ez_x, int size_Ez_y, 
					int ele_ex, int ele_hx, int ele_hy
)
{
	int i, j;
	float dif_Hy, dif_Hx;
	for (i = 1; i < size_Ez_y - 1; i++){
		for (j = 1; j < size_Ez_x - 1; j++){
			//Hy(i,j)	-	Hy(i-1,j)
			dif_Hy = Hy[i*ele_hy + j] - Hy[(i - 1)* ele_hy + j];
			//Hx(i,j-1)	-	Hx(i,j)
			dif_Hx = Hx[i*ele_hx + (j - 1)] - Hx[i*ele_hx + j];
			Ez[i*ele_hx + j] += coe_Ez * (dif_Hx + dif_Hy);
		}
	}
}

__global__ void Hx_cmp_kernel(
	float* Ez, float* Hx, 
	int size_Hx_x, int size_Hx_y,
	float coe_H, int ele_ex, int ele_hx
)
{
	int i, j;

	for ( i = 0; i < size_Hx_y; i++){
		for ( j = 0; j < size_Hx_x; j++){
			Hx[i*ele_hx + j] += coe_H *
				(Ez[i*ele_ex + j] - Ez[i*ele_ex + (j + 1)]);
		}
	}
}

__global__ void Hy_cmp_kernel(
	float* Ez, float* Hy,
	int size_Hy_x, int size_Hy_y,
	float coe_H, int ele_ex, int ele_hy
)
{
	int i, j;

	for ( i = 0; i < size_Hy_y; i++){
		for ( j = 0; j < size_Hy_x; j++){
			Hy[i*ele_hy + j] += 
				coe_H * (Ez[(i + 1)*ele_ex + j] - Ez[i*ele_ex + j]);
		}
	}
}

__global__ void Ez_boundary_PEC(
	float* Ez, 
	int size_Ez_x, int size_Ez_y, int ele_ex
	)
{
	int i, j;

	for (i = 0; i < size_Ez_y; i++){
		if (i == 0 || i == (size_Ez_y - 1)){
			for (j = 0; j < size_Ez_x; j++){
				Ez[i * ele_ex + j] = 0.f;
			}
		}
		else{
			Ez[i* ele_ex + 0] = 0.f;
			Ez[i * ele_ex + size_Ez_x - 1] = 0.f;
		}
	}
}

