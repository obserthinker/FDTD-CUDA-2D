#include "hip/hip_runtime.h"
#include "cmp.cuh"

__global__
void src_cmp_kernel(
int current_timestep,
float* dev_Ez,
int size_Ez_x, int size_Ez_y, float dt, int ele_ex
)
{
	float T, T0;
	float vt, val_src, time;
	int src_pos_x, src_pos_y;

	time = current_timestep * dt;

	T = 5e-10f;
	T0 = 3 * T;
	vt = (time - T0) / T;

	val_src = expf(-powf(vt, 2.0f));

	src_pos_x = size_Ez_y / 2;
	src_pos_y = size_Ez_x / 2;

	dev_Ez[src_pos_y * ele_ex + src_pos_x] = val_src;
}

__global__
void Ez_cmp_kernel(
float* Ez, float* Hx, float* Hy,
float coe_Ez, int size_Ez_x, int size_Ez_y,
int ele_ex, int ele_hx, int ele_hy
)
{
	int x, y;
	int tid, number;
	tid = threadIdx.x + blockIdx.x*blockDim.x;
	float dif_Hy, dif_Hx;
	while (tid < ele_ex*size_Ez_y)
	{
		number = tid + 1;
		y = number % ele_ex;//row
		x = number - (y*ele_ex);//column
		//Hy(i,j)	-	Hy(i-1,j)
		dif_Hy = Hy[y*ele_hy + x] - Hy[(y - 1)* ele_hy + x];
		//Hx(i,j-1)	-	Hx(i,j)
		dif_Hx = Hx[y*ele_hx + (x - 1)] - Hx[y*ele_hx + x];
		Ez[y*ele_ex + x] += coe_Ez * (dif_Hx + dif_Hy);
		tid += blockDim.x*gridDim.x;
	}
}

__global__ void Hx_cmp_kernel(
	float* Ez, float* Hx,
	int size_Hx_x, int size_Hx_y,
	float coe_H, int ele_ex, int ele_hx
	)
{
	int x, y;
	int tid, number;
	tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < ele_hx*size_Hx_y)
	{
		number = tid + 1;
		y = number % ele_hx;
		x = number - (y*ele_hx);
		Hx[y*ele_hx + x] += coe_H *(Ez[y*ele_ex + x] - Ez[y*ele_ex + (x + 1)]);
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void Hy_cmp_kernel(
	float* Ez, float* Hy,
	int size_Hy_x, int size_Hy_y,
	float coe_H, int ele_ex, int ele_hy
	)
{
	int x, y;
	int tid, number;
	tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < ele_hy * size_Hy_y)
	{
		number = tid + 1;
		y = number % ele_hy;
		x = number - (y*ele_hy);
		Hy[y*ele_hy + x] += coe_H * (Ez[(y + 1)*ele_ex + x] - Ez[y*ele_ex + x]);
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void Ez_boundary_PEC(
	float* Ez,
	int size_Ez_x, int size_Ez_y, int ele_ex
	)
{
	int i, j;

	for (i = 0; i < size_Ez_y; i++){
		if (i == 0 || i == (size_Ez_y - 1)){
			for (j = 0; j < size_Ez_x; j++){
				Ez[i * ele_ex + j] = 0.f;
			}
		}
		else{
			Ez[i* ele_ex + 0] = 0.f;
			Ez[i * ele_ex + size_Ez_x - 1] = 0.f;
		}
	}
}

__global__
void Ez_MUR_u(
float* Ez, float* E_bd_u, float* E_nbd_u,
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	int tid;
	tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < size_Ez_x){
		Ez[(size_Ez_y - 1) * ele_ex + tid] = E_nbd_u[tid]
			+ coe_MUR * (Ez[(size_Ez_y - 2) * ele_ex + tid]
			- E_bd_u[tid]);
		E_nbd_u[tid] = Ez[(size_Ez_y - 2) * ele_ex + tid];
		E_bd_u[tid] = Ez[(size_Ez_y - 1) * ele_ex + tid];
		tid += blockDim.x * gridDim.x;
	}
}

__global__
void Ez_MUR_d(
float* Ez, float* E_bd_d, float* E_nbd_d,
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	int tid;
	tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < size_Ez_x)
	{
		Ez[tid] = E_nbd_d[tid] + coe_MUR * (Ez[1 * ele_ex + tid]
			- E_bd_d[tid]);
		E_nbd_d[tid] = Ez[1 * ele_ex + tid];
		E_bd_d[tid] = Ez[tid];
		tid += blockDim.x * gridDim.x;
	}
}

__global__
void Ez_MUR_lr(
float* Ez, float* E_bd_l, float* E_nbd_l,
float* E_bd_r, float* E_nbd_r,
int size_Ez_x, int size_Ez_y, float coe_MUR, int ele_ex
)
{
	int tid;
	tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < size_Ez_y)
	{
		//left
		Ez[tid*ele_ex + 0] = E_nbd_l[tid] + coe_MUR *
			(Ez[tid * ele_ex + 1] - E_bd_l[tid]);
		E_nbd_l[tid] = Ez[tid * ele_ex + 1];
		E_bd_l[tid] = Ez[tid * ele_ex + 0];
		//right
		Ez[tid* ele_ex + (size_Ez_x - 1)] = E_nbd_r[tid] + coe_MUR *
			(Ez[tid * ele_ex + (size_Ez_x - 2)] - E_bd_r[tid]);
		E_nbd_r[tid] = Ez[tid * ele_ex + (size_Ez_x - 2)];
		E_bd_r[tid] = Ez[tid* ele_ex + (size_Ez_x - 1)];

		tid += blockDim.x * gridDim.x;
	}
}
